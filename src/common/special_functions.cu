#include "special_functions.h"
#include <hip/hip_runtime.h>
#include <map>
#include <algorithm>

namespace fmm {

namespace detail {

    void cudaCopyMathConstants()
    {
#ifndef FMM_CONSTEXPR_MATH
        hipMalloc(&dev_Knm, Knm.Knm.size() * sizeof(double));
        hipMemcpy(dev_Knm, Knm.Knm.data(), Knm.Knm.size() * sizeof(double), hipMemcpyHostToDevice);
        hipMalloc(&dev_Anm, Anm.Anm.size() * sizeof(double));
        hipMemcpy(dev_Anm, Anm.Anm.data(), Anm.Anm.size() * sizeof(double), hipMemcpyHostToDevice);
        hipMalloc(&dev_m2lcoef, m2lcoef.m2lcoef.size() * sizeof(double));
        hipMemcpy(dev_m2lcoef, m2lcoef.m2lcoef.data(), m2lcoef.m2lcoef.size() * sizeof(double), hipMemcpyHostToDevice);
#endif // !FMM_CONSTEXPR_MATH

        size_t dm_size = dmatrix.size();
        hipMalloc(&dev_dmatrix, dm_size * dmatrix.begin()->second.size() * sizeof(double));
        hipMalloc(&dev_dm_map, dm_size * sizeof(int));

        std::vector<int> keys(dm_size), keymap(dm_size);
        for (int i = 0; const auto& [key, value] : dmatrix)
        {
            keys[i++] = key;
        }
        std::sort(keys.begin(), keys.end());
        for (int i = 0; i < dm_size; ++i)
        {
            keymap[i] = keys[i];
        }
        hipMemcpy(dev_dm_map, keymap.data(), dm_size * sizeof(int), hipMemcpyHostToDevice);
        for (int i = 0; const auto& key : keys)
        {
            const auto& value = dmatrix[key];
            hipMemcpy(dev_dmatrix + i * value.size(), value.data(), value.size() * sizeof(double), hipMemcpyHostToDevice);
            ++i;
        }
    }

    void cudaClearMathConstants()
    {
#ifndef FMM_CONSTEXPR_MATH
        hipFree(&dev_Knm);
        hipFree(&dev_Anm);
        hipFree(&dev_m2lcoef);
#endif
        hipFree(&dev_dm_map);
        hipFree(&dev_dmatrix);
    }

} // detail

} // fmm